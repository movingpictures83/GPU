#include <gputk.h>

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ INSERT CODE HERE

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  gpuTKImage_t inputImage;
  gpuTKImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = gpuTKArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = gpuTKArg_getInputFile(args, 0);

  inputImage = gpuTKImport(inputImageFile);

  imageWidth  = gpuTKImage_getWidth(inputImage);
  imageHeight = gpuTKImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = gpuTKImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = gpuTKImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = gpuTKImage_getData(inputImage);
  hostOutputImageData = gpuTKImage_getData(outputImage);

  gpuTKTime_start(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  gpuTKTime_stop(GPU, "Doing GPU memory allocation");

  gpuTKTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  gpuTKTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  gpuTKTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE

  gpuTKTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  gpuTKTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  gpuTKTime_stop(Copy, "Copying data from the GPU");

  gpuTKTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  gpuTKImage_delete(outputImage);
  gpuTKImage_delete(inputImage);

  return 0;
}
