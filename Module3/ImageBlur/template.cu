#include <gputk.h>

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      gpuTKLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5

//@@ INSERT CODE HERE

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  gpuTKImage_t inputImage;
  gpuTKImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = gpuTKArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = gpuTKArg_getInputFile(args, 0);

  inputImage = gpuTKImport(inputImageFile);

  // The input image is in grayscale, so the number of channels
  // is 1
  imageWidth  = gpuTKImage_getWidth(inputImage);
  imageHeight = gpuTKImage_getHeight(inputImage);

  // Since the image is monochromatic, it only contains only one channel
  outputImage = gpuTKImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = gpuTKImage_getData(inputImage);
  hostOutputImageData = gpuTKImage_getData(outputImage);

  gpuTKTime_start(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  gpuTKTime_stop(GPU, "Doing GPU memory allocation");

  gpuTKTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyHostToDevice);
  gpuTKTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  gpuTKTime_start(Compute, "Doing the computation on the GPU");

  gpuTKTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  gpuTKTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  gpuTKTime_stop(Copy, "Copying data from the GPU");

  gpuTKTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  gpuTKSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  gpuTKImage_delete(outputImage);
  gpuTKImage_delete(inputImage);

  return 0;
}
