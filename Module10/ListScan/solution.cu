#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// + lst[n-1]}

#include <gputk.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define gpuTKCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void fixup(float *input, float *aux, int len) {
  unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
  if (blockIdx.x) {
    if (start + t < len)
      input[start + t] += aux[blockIdx.x - 1];
    if (start + BLOCK_SIZE + t < len)
      input[start + BLOCK_SIZE + t] += aux[blockIdx.x - 1];
  }
}

__global__ void scan(float *input, float *output, float *aux, int len) {
  // Load a segment of the input vector into shared memory
  __shared__ float scan_array[BLOCK_SIZE << 1];
  unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
  if (start + t < len)
    scan_array[t] = input[start + t];
  else
    scan_array[t] = 0;
  if (start + BLOCK_SIZE + t < len)
    scan_array[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
  else
    scan_array[BLOCK_SIZE + t] = 0;
  __syncthreads();

  // Reduction
  int stride;
  for (stride = 1; stride <= BLOCK_SIZE; stride <<= 1) {
    int index = (t + 1) * stride * 2 - 1;
    if (index < 2 * BLOCK_SIZE)
      scan_array[index] += scan_array[index - stride];
    __syncthreads();
  }

  // Post reduction
  for (stride = BLOCK_SIZE >> 1; stride; stride >>= 1) {
    int index = (t + 1) * stride * 2 - 1;
    if (index + stride < 2 * BLOCK_SIZE)
      scan_array[index + stride] += scan_array[index];
    __syncthreads();
  }

  if (start + t < len)
    output[start + t] = scan_array[t];
  if (start + BLOCK_SIZE + t < len)
    output[start + BLOCK_SIZE + t] = scan_array[BLOCK_SIZE + t];

  if (aux && t == 0)
    aux[blockIdx.x] = scan_array[2 * BLOCK_SIZE - 1];
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  float *deviceAuxArray, *deviceAuxScannedArray;
  int numElements; // number of elements in the list

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &numElements);
  hipHostAlloc(&hostOutput, numElements * sizeof(float),
                hipHostMallocDefault);
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The number of input elements in the input is ",
        numElements);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  gpuTKCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  gpuTKCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));

  // XXX the size is fixed for ease of implementation.
  hipMalloc(&deviceAuxArray, (BLOCK_SIZE << 1) * sizeof(float));
  hipMalloc(&deviceAuxScannedArray, (BLOCK_SIZE << 1) * sizeof(float));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Clearing output memory.");
  gpuTKCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  gpuTKTime_stop(GPU, "Clearing output memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  gpuTKCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  int numBlocks = ceil((float)numElements / (BLOCK_SIZE << 1));
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  gpuTKLog(TRACE, "The number of blocks is ", numBlocks);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, deviceAuxArray,
                              numElements);
  hipDeviceSynchronize();
  scan<<<dim3(1, 1, 1), dimBlock>>>(deviceAuxArray, deviceAuxScannedArray,
                                    NULL, BLOCK_SIZE << 1);
  hipDeviceSynchronize();
  fixup<<<dimGrid, dimBlock>>>(deviceOutput, deviceAuxScannedArray,
                               numElements);

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  gpuTKCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipFree(deviceAuxArray);
  hipFree(deviceAuxScannedArray);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, numElements);

  free(hostInput);
  hipHostFree(hostOutput);

  return 0;
}
