#include "hip/hip_runtime.h"
#include <gputk.h>

#define NUM_BINS 128

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void histogram_kernel(const char *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Privatized bins
  extern __shared__ unsigned int bins_s[];
  for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
       binIdx += blockDim.x) {
    bins_s[binIdx] = 0;
  }
  __syncthreads();

  // Histogram
  for (unsigned int i = tid; i < num_elements;
       i += blockDim.x * gridDim.x) {
    atomicAdd(&(bins_s[(unsigned int)input[i]]), 1);
  }
  __syncthreads();

  // Commit to global memory
  for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
       binIdx += blockDim.x) {
    atomicAdd(&(bins[binIdx]), bins_s[binIdx]);
  }
}

void histogram(const char *input, unsigned int *bins,
               unsigned int num_elements, unsigned int num_bins) {

  // zero out bins
  CUDA_CHECK(hipMemset(bins, 0, num_bins * sizeof(unsigned int)));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(256), gridDim(30);
    histogram_kernel<<<gridDim, blockDim,
                       num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int inputLength;
  char *hostInput;
  unsigned int *hostBins;
  char *deviceInput;
  unsigned int *deviceBins;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (char *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength, "Text");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);
  gpuTKLog(TRACE, "The number of bins is ", NUM_BINS);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  CUDA_CHECK(hipMalloc((void **)&deviceInput, inputLength));
  CUDA_CHECK(
      hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceInput, hostInput, inputLength,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  gpuTKLog(TRACE, "Launching kernel");
  gpuTKTime_start(Compute, "Performing CUDA computation");
  // @@ Insert code here
  histogram(deviceInput, deviceBins, inputLength, NUM_BINS);
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  CUDA_CHECK(hipFree(deviceInput));
  CUDA_CHECK(hipFree(deviceBins));
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  gpuTKSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
