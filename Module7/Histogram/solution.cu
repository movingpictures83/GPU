#include "hip/hip_runtime.h"
#include <gputk.h>

#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Privatized bins
  extern __shared__ unsigned int bins_s[];
  for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
       binIdx += blockDim.x) {
    bins_s[binIdx] = 0;
  }
  __syncthreads();

  // Histogram
  for (unsigned int i = tid; i < num_elements;
       i += blockDim.x * gridDim.x) {
    atomicAdd(&(bins_s[input[i]]), 1);
  }
  __syncthreads();

  // Commit to global memory
  for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
       binIdx += blockDim.x) {
    atomicAdd(&(bins[binIdx]), bins_s[binIdx]);
  }
}

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num_bins) {
    bins[tid] = min(bins[tid], 127);
  }
}

void histogram(unsigned int *input, unsigned int *bins,
               unsigned int num_elements, unsigned int num_bins) {

  // zero out bins
  CUDA_CHECK(hipMemset(bins, 0, num_bins * sizeof(unsigned int)));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(512), gridDim(30);
    histogram_kernel<<<gridDim, blockDim,
                       num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  // Make sure bin values are not too large
  {
    dim3 blockDim(512);
    dim3 gridDim((num_bins + blockDim.x - 1) / blockDim.x);
    convert_kernel<<<gridDim, blockDim>>>(bins, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

int main(int argc, char *argv[]) {
  gpuTKArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)gpuTKImport(gpuTKArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);
  gpuTKLog(TRACE, "The number of bins is ", NUM_BINS);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  CUDA_CHECK(hipMalloc((void **)&deviceInput,
                        inputLength * sizeof(unsigned int)));
  CUDA_CHECK(
      hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceInput, hostInput,
                        inputLength * sizeof(unsigned int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  gpuTKLog(TRACE, "Launching kernel");
  gpuTKTime_start(Compute, "Performing CUDA computation");

  histogram(deviceInput, deviceBins, inputLength, NUM_BINS);
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  CUDA_CHECK(hipFree(deviceInput));
  CUDA_CHECK(hipFree(deviceBins));
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  gpuTKSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
