#include "hip/hip_runtime.h"
#include <gputk.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>

#define gpuTKCheck(stmt)                                                  \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      gpuTKLog(ERROR, "Failed to run stmt ", #stmt);                      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Data type to use for the matrices
typedef double real_t;

// Initialize a matrix A with random values between min and max
void matrix_init(real_t *A, int rows, int cols, real_t min=0., real_t max=1.) {
	for(int i = 0; i < (rows * cols); ++i)
		A[i] = (max - min) * (rand() % 51 ) / 50. + min;
}

// Verify the result of C = A * B
double verify(real_t *C, real_t *A, real_t *B, int CRows, int CCols, int ACols) {
	double error = 0;
#pragma omp parallel for collapse(2)
	for(int i = 0; i < CRows; ++i) {
		for(int j = 0; j < CCols; ++j) {
			real_t Cij = 0;
			for(int k = 0; k < ACols; ++k)
				Cij += A[i * ACols + k] * B[k * CCols + j];
			error += abs(Cij - C[i * CCols + j]);
		}
	}
	return error;
}

// Compute C = A * B
template <int TILE_SIZE=32> __global__ void matrixMultiply(real_t *C,
		real_t *A, real_t *B, int CRows, int CCols, int ACols) {
	//@@ Insert code to implement matrix multiplication here
	__shared__ real_t As[TILE_SIZE][TILE_SIZE];
	__shared__ real_t Bs[TILE_SIZE][TILE_SIZE];
	int ti = threadIdx.y;
	int tj = threadIdx.x;
	int i = blockIdx.y * TILE_SIZE + ti;
	int j = blockIdx.x * TILE_SIZE + tj;
	real_t Cij=0;
	for(int k = 0; k < (ACols+TILE_SIZE-1)/TILE_SIZE; ++k) {
		As[ti][tj] = (i < CRows) && ((k * TILE_SIZE + tj) < ACols) ?
				A[i * ACols + k * TILE_SIZE + tj] : 0;
		Bs[ti][tj] = (j < CCols) && ((k * TILE_SIZE + ti) < ACols) ?
				B[(k * TILE_SIZE + ti) * CCols + j] : 0;
		__syncthreads();
#pragma unroll
		for(int tk = 0; tk < TILE_SIZE; ++tk)
			Cij += As[ti][tk] * Bs[tk][tj];
		__syncthreads();
	}
	if((i < CRows) && (j < CCols))
		C[i * CCols + j]=Cij;
}

int main(int argc, char **argv) {
	constexpr int TILE_SIZE=32;
	gpuTKArg_t args;
	real_t *A; // The A matrix
	real_t *B; // The B matrix
	real_t *C; // The output matrix
	int ARows; // number of rows in the matrix A
	int ACols; // number of cols in the matrix A
	int BRows; // number of rows in the matrix B
	int BCols; // number of cols in the matrix B
	int CRows; // number of rows in the matrix C
	int CCols; // number of cols in the matrix C
	int concurrentAccessQ = 0;
	int device; // current device
	hipStream_t stream;  // stream to run the computations

	// Get current device
	gpuTKCheck(hipGetDevice(&device));
	// Get hipDeviceAttributeConcurrentManagedAccess device property
	gpuTKCheck(hipDeviceGetAttribute(&concurrentAccessQ,
			hipDeviceAttributeConcurrentManagedAccess,device));
	// Create stream
	gpuTKCheck(hipStreamCreate(&stream));

	args = gpuTKArg_read(argc, argv);
	// Read matrices input sizes
	ARows = atoi(gpuTKArg_getInputFile(args, 0));
	ACols = atoi(gpuTKArg_getInputFile(args, 1));
	BCols = atoi(gpuTKArg_getInputFile(args, 2));

	// Set CRows, CCols and BRows
	CRows = ARows;
	CCols = BCols;
	BRows = ACols;

	gpuTKLog(TRACE, "The dimensions of A are ", ARows, " x ", ACols);
	gpuTKLog(TRACE, "The dimensions of B are ", BRows, " x ", BCols);
	gpuTKLog(TRACE, "The dimensions of C are ", CRows, " x ", CCols);

	gpuTKTime_start(GPU, "Allocating Managed Memory");
	//@@ Insert code to allocate magaed memory here
	// Allocate the A matrix
	gpuTKCheck(hipMallocManaged((void**) &A,
			CRows * ACols * sizeof(real_t)));
	// Allocate the B matrix
	gpuTKCheck(hipMallocManaged((void**) &B,
			ACols * CCols * sizeof(real_t)));
	// Allocate the C matrix
	gpuTKCheck(hipMallocManaged((void**) &C,
			CRows * CCols * sizeof(real_t)));
	gpuTKTime_stop(GPU, "Allocating Managed Memory");

	gpuTKTime_start(GPU, "Prefetching and advising Managed Memory");
	//@@ Insert code to prefetch data and set advises here
	// Setting memory advise to matrices A and B
	gpuTKCheck(hipMemAdvise(A, CRows * ACols * sizeof(real_t),
			hipMemAdviseSetReadMostly, hipCpuDeviceId));
	gpuTKCheck(hipMemAdvise(B, ACols * CCols * sizeof(real_t),
			hipMemAdviseSetReadMostly, hipCpuDeviceId));
	if(concurrentAccessQ != 0) {
		//@@ Insert code to prefetch data here
		//@@ Prefetch matrices to the host
		gpuTKCheck(hipMemPrefetchAsync(A,
				CRows * ACols * sizeof(real_t), hipCpuDeviceId));
		gpuTKCheck(hipMemPrefetchAsync(B,
				ACols * CCols * sizeof(real_t), hipCpuDeviceId));
	}
	gpuTKTime_stop(GPU, "Prefetching and advising Managed Memory");

	gpuTKTime_start(Generic, "Initializing memory on host");
	// Initialize matrices A and B with random numbers
	matrix_init(A, CRows, ACols);
	matrix_init(B, ACols, CCols);
	gpuTKTime_stop(Generic, "Initializing memory on host");

	gpuTKTime_start(CopyAsync, "Prefetching GPU memory to device");
	if(concurrentAccessQ!=0) {
		//@@ Insert code to prefetch data here
		//@@ Prefetch matrices to the device
		gpuTKCheck(hipMemPrefetchAsync(A, CRows * ACols * sizeof(real_t),
				device, stream));
		gpuTKCheck(hipMemPrefetchAsync(B, ACols * CCols * sizeof(real_t),
				device, stream));
		gpuTKCheck(hipMemPrefetchAsync(C, CRows * CCols * sizeof(real_t),
				device, stream));
	}
	gpuTKTime_stop(CopyAsync, "Prefetching GPU memory to device");

	gpuTKTime_start(Compute, "Performing CUDA computation");
	//@@ Initialize the grid and block dimensions here
	dim3 threads(TILE_SIZE, TILE_SIZE);
	dim3 grid((CCols+threads.x-1) / threads.x,
			(CRows+threads.y-1) / threads.y);
	//@@ Launch the GPU Kernel here
	// Perform the matrix multiplication
	matrixMultiply<TILE_SIZE><<<grid, threads, 0, stream>>>(C, A, B, CRows,
			CCols, ACols);
	gpuTKCheck(hipDeviceSynchronize());
	gpuTKTime_stop(Compute, "Performing CUDA computation");


	gpuTKTime_start(CopyAsync, "Prefetching GPU memory to device");
	if(concurrentAccessQ!=0) {
		//@@ Insert code to prefetch data here
		// Prefetch the ouput matrix to the host
		gpuTKCheck(hipMemPrefetchAsync(C, CRows * CCols * sizeof(real_t),
				hipCpuDeviceId, stream));
	}
	gpuTKTime_stop(CopyAsync, "Prefetching GPU memory to host");

	gpuTKTime_start(Generic, "Verifying matrix multiplication result");
	// Computing numeric error of the computation
	// The error should be somewhat less than 10E-6
	gpuTKLog(TRACE, "Numeric error: ", verify(C, A, B, CRows, CCols, ACols));
	gpuTKTime_stop(Generic, "Verifying matrix multiplication result");

	gpuTKTime_start(GPU, "Freeing Managed Memory");
	//@@ Free the GPU memory here
	// Freeing memory
	gpuTKCheck(hipFree(A));
	gpuTKCheck(hipFree(B));
	gpuTKCheck(hipFree(C));
	gpuTKTime_stop(GPU, "Freeing Managed Memory");
	return 0;
}
