#include <gputk.h>

//@@ The purpose of this code is to become familiar with the submission
//@@ process. Do not worry if you do not understand all the details of
//@@ the code.

int main(int argc, char **argv) {
  int deviceCount;

  gpuTKArg_read(argc, argv);

  hipGetDeviceCount(&deviceCount);

  gpuTKTime_start(GPU, "Getting GPU Data."); //@@ start a timer

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        gpuTKLog(TRACE, "No CUDA GPU has been detected");
        return -1;
      } else if (deviceCount == 1) {
        //@@ gpuTKLog is a provided logging API (similar to Log4J).
        //@@ The logging function gpuTKLog takes a level which is either
        //@@ OFF, FATAL, ERROR, WARN, INFO, DEBUG, or TRACE and a
        //@@ message to be printed.
        gpuTKLog(TRACE, "There is 1 device supporting CUDA");
      } else {
        gpuTKLog(TRACE, "There are ", deviceCount,
              " devices supporting CUDA");
      }
    }

    gpuTKLog(TRACE, "Device ", dev, " name: ", deviceProp.name);
    gpuTKLog(TRACE, " Computational Capabilities: ", deviceProp.major, ".",
          deviceProp.minor);
    gpuTKLog(TRACE, " Maximum global memory size: ",
          deviceProp.totalGlobalMem);
    gpuTKLog(TRACE, " Maximum constant memory size: ",
          deviceProp.totalConstMem);
    gpuTKLog(TRACE, " Maximum shared memory size per block: ",
          deviceProp.sharedMemPerBlock);
    gpuTKLog(TRACE, " Maximum block dimensions: ",
          deviceProp.maxThreadsDim[0], " x ", deviceProp.maxThreadsDim[1],
          " x ", deviceProp.maxThreadsDim[2]);
    gpuTKLog(TRACE, " Maximum grid dimensions: ", deviceProp.maxGridSize[0],
          " x ", deviceProp.maxGridSize[1], " x ",
          deviceProp.maxGridSize[2]);
    gpuTKLog(TRACE, " Warp size: ", deviceProp.warpSize);
  }

  gpuTKTime_stop(GPU, "Getting GPU Data."); //@@ stop the timer

  return 0;
}
